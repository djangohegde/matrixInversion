#include "hip/hip_runtime.h"
#include "luShared.cuh"

int main() {
    FpType *A, *L, *U, *LU;
    FpType *d_A, *d_L, *d_U;
    int matrixSize, numThreads;
    // let user input matrixSize and numThreads
    printf("Enter matrix size: ");
    scanf("%d", &matrixSize);
    printf("Enter number of threads: ");
    scanf("%d", &numThreads);
    int i=0, j=0;
    int numElements = matrixSize * matrixSize;
    FpType startT=0.0, endT=0.0;

    A = (FpType *)malloc(numElements * sizeof(FpType));
    L = (FpType *)malloc(numElements * sizeof(FpType));
    U = (FpType *)malloc(numElements * sizeof(FpType));
    LU = (FpType *)malloc(numElements * sizeof(FpType));

    hipMalloc(&d_A, numElements * sizeof(FpType));
    hipMalloc(&d_L, numElements * sizeof(FpType));
    hipMalloc(&d_U, numElements * sizeof(FpType));

    // FpType inputMatrix[] = {4, 11, 3, 4, 10, 4, 2, 4, 2};

    FILE *file = fopen("matrix.txt", "r");

    srand(time(NULL));
    for (i = 0; i < matrixSize; i++) {
        for (j = 0; j < matrixSize; j++) {
            fscanf(file, "%lf", &A[i * matrixSize + j]);
            // A[i * matrixSize + j] = inputMatrix[i * matrixSize + j];
            // A[i * matrixSize + j] = rand() % 10 + 1;
            L[i * matrixSize + j] = 0.0;
            U[i * matrixSize + j] = 0.0;
        }
    }

    hipMemcpy(d_A, A, numElements * sizeof(FpType), hipMemcpyHostToDevice);
    hipMemcpy(d_L, L, numElements * sizeof(FpType), hipMemcpyHostToDevice);
    hipMemcpy(d_U, U, numElements * sizeof(FpType), hipMemcpyHostToDevice);

    int shMemSize = 3 * matrixSize * matrixSize * sizeof(FpType);

    startT = clock();
    lu_decomp<<<1, numThreads, shMemSize>>>(d_A, d_L, d_U, matrixSize);
    endT = clock();

    hipMemcpy(A, d_A, numElements * sizeof(FpType), hipMemcpyDeviceToHost);
    hipMemcpy(L, d_L, numElements * sizeof(FpType), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, numElements * sizeof(FpType), hipMemcpyDeviceToHost);

    printf("Time taken: %f\n", (endT - startT) / CLOCKS_PER_SEC);

    // // print A
    // printf("Orig A:\n");
    // for (i = 0; i < matrixSize; i++) {
    //     for (j = 0; j < matrixSize; j++) {
    //         printf("%f ", A[i * matrixSize + j]);
    //     }
    //     printf("\n");
    // }

    // // print U
    // printf("U:\n");
    // for (i = 0; i < matrixSize; i++) {
    //     for (j = 0; j < matrixSize; j++) {
    //         printf("%f ", U[i * matrixSize + j]);
    //     }
    //     printf("\n");
    // }

    // // print L
    // printf("L:\n");
    // for (i = 0; i < matrixSize; i++) {
    //     for (j = 0; j < matrixSize; j++) {
    //         printf("%f ", L[i * matrixSize + j]);
    //     }
    //     printf("\n");
    // }

    // multipy L and U to check if A = LU
    for (i = 0; i < matrixSize; i++) {
        for (j = 0; j < matrixSize; j++) {
            FpType sum = 0.0;
            for (int k = 0; k < matrixSize; k++) {
                sum += L[i * matrixSize + k] * U[k * matrixSize + j];
                LU[i * matrixSize + j] = sum;
            }
        }
    }

    FpType diff=0.0;

    // subtract A from LU
    for (i = 0; i < matrixSize; i++) {
        for (j = 0; j < matrixSize; j++) {
            diff += fabs(LU[i * matrixSize + j] - A[i * matrixSize + j]);
        }
    }

    if (fabs(diff) < 10e-6) {
        printf("LU decomposition is correct\n");
    } else {
        printf("LU decomposition is incorrect\n");
    }

    printf("Diff is %f\n", diff);

    hipFree(d_A);
    hipFree(d_L);
    hipFree(d_U);

    free(A);
    free(L);
    free(U);
    free(LU);

    return 0;
}
