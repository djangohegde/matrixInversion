#include "hip/hip_runtime.h"
#include "luBatched.cuh"

int main() {
    int matrixSize, numMatrices, numThreads;

    // let user input matrix size and number of matrices
    printf("Enter matrix size: ");
    scanf("%d", &matrixSize);
    printf("Enter number of matrices: ");
    scanf("%d", &numMatrices);
    printf("Enter number of threads: ");
    scanf("%d", &numThreads);
    
    FpType *A, *L, *U, *LU, *LandU;
    FpType *d_A, *d_L, *d_U;
    int i=0, j=0;
    int numElements = matrixSize * matrixSize * numMatrices;
    FpType startT=0.0, endT=0.0;

    A = (FpType *)malloc(numElements * sizeof(FpType));
    L = (FpType *)malloc(numElements * sizeof(FpType));
    U = (FpType *)malloc(numElements * sizeof(FpType));
    LU = (FpType *)malloc(matrixSize * matrixSize * sizeof(FpType));
    LandU = (FpType *)malloc(numElements * sizeof(FpType));

    hipMalloc(&d_A, numElements * sizeof(FpType));
    hipMalloc(&d_L, numElements * sizeof(FpType));
    hipMalloc(&d_U, numElements * sizeof(FpType));

    FpType inputMatrix[] = {4, 11, 3, 4, 10, 4, 2, 4, 2};

    FILE *f; // = fopen("matrix.txt", "r");

    for (int k = 0; k < numMatrices; k++) {
        int offset = k * matrixSize * numMatrices;
        f = fopen("matrix.txt", "r");
        for (i = 0; i < matrixSize; i++) {
            for (j = 0; j < matrixSize; j++) {
                // fscanf(f, "%lf", &A[(i * matrixSize) + offset + j]);
                A[(i * matrixSize) + offset + j] = inputMatrix[i * matrixSize + j];
                // A[(i * matrixSize) + offset + j] = rand() % 10 + 1;
                L[(i * matrixSize) + offset + j] = 0.0;
                U[(i * matrixSize) + offset + j] = 0.0;
                LandU[(i * matrixSize) + offset + j] = 0.0;
            }
        }
    }

    hipMemcpy(d_A, A, numElements * sizeof(FpType), hipMemcpyHostToDevice);
    hipMemcpy(d_L, L, numElements * sizeof(FpType), hipMemcpyHostToDevice);
    hipMemcpy(d_U, U, numElements * sizeof(FpType), hipMemcpyHostToDevice);

    int shMemSize = matrixSize * matrixSize * sizeof(FpType);

    startT = clock();
    batched_lu<<<numMatrices, numThreads>>>(d_A, d_L, d_U, matrixSize, numMatrices);
    endT = clock();

    hipMemcpy(A, d_A, numElements * sizeof(FpType), hipMemcpyDeviceToHost);
    hipMemcpy(L, d_L, numElements * sizeof(FpType), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, numElements * sizeof(FpType), hipMemcpyDeviceToHost);

    // // assign L and U
    // for (int k = 0; k < numMatrices; k++) {
    //     int offset = k * matrixSize * numMatrices;
    //     for (i = 0; i < matrixSize; i++) {
    //         for (j = 0; j < matrixSize; j++) {
    //             if (i > j) {
    //                 L[(i * matrixSize) + offset + j] = LandU[(i * matrixSize) + offset + j];
    //                 U[(i * matrixSize) + offset + j] = 0.0;
    //             }
    //             else if (i == j) {
    //                 L[(i * matrixSize) + offset + j] = 1.0;
    //                 U[(i * matrixSize) + offset + j] = LandU[(i * matrixSize) + offset + j];
    //             }
    //             else {
    //                 L[(i * matrixSize) + offset + j] = 0.0;
    //                 U[(i * matrixSize) + offset + j] = LandU[(i * matrixSize) + offset + j];
    //             }
    //         }
    //     }
    // }

    // // print all L and U
    // for (int k = 0; k < numMatrices; k++) {
    //     int offset = k * matrixSize * numMatrices;
    //     printf("Matrix %d\n", k+1);
    //     printf("L\n");
    //     for (i = 0; i < matrixSize; i++) {
    //         for (j = 0; j < matrixSize; j++) {
    //             printf("%f ", L[(i * matrixSize) + offset + j]);
    //         }
    //         printf("\n");
    //     }

    //     printf("U\n");
    //     for (i = 0; i < matrixSize; i++) {
    //         for (j = 0; j < matrixSize; j++) {
    //             printf("%f ", U[(i * matrixSize) + offset + j]);
    //         }
    //         printf("\n");
    //     }
    // }

    // int k=0, l=0, offset=0, wrngLU=0, corrLU=0, nanLU=0;
    // FpType sum=0.0, diff=0.0;

    // for (k = 0; k < numMatrices; k++) {
    //     for (i = 0; i < matrixSize; i++) {
    //         for (j = 0; j < matrixSize; j++) {
    //             sum = 0.0;
    //             offset = k * numMatrices * matrixSize;
    //             for (l = 0; l < matrixSize; l++) {
    //                 sum += L[(i * matrixSize) + offset + l] * U[(l * matrixSize) + offset + j];
    //                 LU[(i * matrixSize) + j] = sum;
    //             }
    //         }
    //     }

    //     // subtract A from LU
    //     diff = 0.0;
    //     for (i = 0; i < matrixSize; i++) {
    //         for (j = 0; j < matrixSize; j++) {
    //             diff += fabs(LU[i * matrixSize + j] - A[(i * matrixSize) + offset + j]);
    //         }
    //     }

    //     if (isnan(diff))  {
    //         nanLU++;
    //         printf("Diff is nan\n");
    //     }

    //     if (fabs(diff) < 10e-6) {
    //         corrLU++;
    //         printf("Correct diff is %f\n", diff);
    //         // printf("LU decomposition is correct\n");
    //         // printf("A = LU\n");
    //     }
        
    //     if (fabs(diff) > 10e-6) {
    //         wrngLU++;
    //         printf("Incorrect diff is %f\n", diff);
    //         // printf("LU decomposition is incorrect\n");
    //         // printf("A != LU\n");
    //     }

    //     // printf("Diff is %f\n", diff);
    // }

    // printf("Matrix size: %d\n", matrixSize);
    // printf("Number of matrices: %d\n", numMatrices);
    // printf("Number of correct LU decompositions: %d\n", corrLU);
    // printf("Number of incorrect LU decompositions: %d\n", wrngLU);
    // printf("Number of nan LU decompositions: %d\n", nanLU);

    printf("Time taken: %f\n", (endT - startT) / CLOCKS_PER_SEC);

    // print A
    printf("A\n");
    for (int i = 0; i < numMatrices * matrixSize; i++) {
        for (int j = 0; j < matrixSize; j++) {
            printf("%f ", A[i * matrixSize + j]);
        }
        printf("\n");
    }

    // print U
    printf("U\n");
    for (int i = 0; i < numMatrices * matrixSize; i++) {
        for (int j = 0; j < matrixSize; j++) {
            printf("%f ", U[i * matrixSize + j]);
        }
        printf("\n");
    }

    // print L
    printf("L\n");
    for (int i = 0; i < numMatrices * matrixSize; i++) {
        for (int j = 0; j < matrixSize; j++) {
            printf("%f ", L[i * matrixSize + j]);
        }
        printf("\n");
    }

    // save L and U to file
    FILE *fL = fopen("L.txt", "w");
    FILE *fU = fopen("U.txt", "w");

    for (int i = 0; i < numMatrices * matrixSize; i++) {
        for (int j = 0; j < matrixSize; j++) {
            fprintf(fL, "%f ", L[i * matrixSize + j]);
            fprintf(fU, "%f ", U[i * matrixSize + j]);
        }
        fprintf(fL, "\n");
        fprintf(fU, "\n");
    }

    fclose(fL);
    fclose(fU);

    hipFree(d_A);
    hipFree(d_L);
    hipFree(d_U);

    free(A);
    free(L);
    free(U);
    free(LU);
    free(LandU);

    return 0;
}
