#include <iostream>
#include <memory>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include "luBatchedInplace.cuh"

int main() {
    int matrixSize, numMatrices, numThreads;

    // let user input matrix size and number of matrices
    std::cout << "Enter matrix size: ";
    std::cin >> matrixSize;
    std::cout << "Enter number of matrices: ";
    std::cin >> numMatrices;
    std::cout << "Enter number of threads: ";
    std::cin >> numThreads;
    
    int numElements = matrixSize * matrixSize * numMatrices;

    auto A = std::make_unique<std::vector<FpType>>(numElements);
    auto A_inv = std::make_unique<std::vector<FpType>>(numElements);

    FpType* d_A;
    hipMalloc(&d_A, numElements * sizeof(FpType));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    FpType inputMatrix[] = {4, 11, 3, 4, 10, 4, 2, 4, 2};

    for (int k = 0; k < numMatrices; ++k) {
        int offset = k * matrixSize * matrixSize;
        for (int i = 0; i < matrixSize; ++i) {
            for (int j = 0; j < matrixSize; ++j) {
                (*A)[(i * matrixSize) + offset + j] = inputMatrix[(i * matrixSize) + j];
                (*A_inv)[(i * matrixSize) + offset + j] = 0.0;
            }
        }
    }

    hipMemcpy(d_A, A->data(), numElements * sizeof(FpType), hipMemcpyHostToDevice);

    int shMemSize = matrixSize * matrixSize * sizeof(FpType);

    hipEventRecord(start, 0);
    batched_lu<<<numMatrices, numThreads, shMemSize>>>(d_A, matrixSize, numMatrices);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " milliseconds\n";

    hipMemcpy(A_inv->data(), d_A, numElements * sizeof(FpType), hipMemcpyDeviceToHost);

    // print A_inv
    for (int k = 0; k < numMatrices; ++k) {
        int offset = k * matrixSize * matrixSize;
        std::cout << "Matrix " << k + 1 << '\n';
        for (int i = 0; i < matrixSize; ++i) {
            for (int j = 0; j < matrixSize; ++j) {
                std::cout << (*A_inv)[(i * matrixSize) + offset + j] << ' ';
            }
            std::cout << '\n';
        }
    }

    // auto startT = std::chrono::high_resolution_clock::now();
    // verifyInv(A->data(), A_inv->data(), matrixSize, numMatrices);
    // auto endT = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> elapsed = endT - startT;
    // std::cout << "Time taken to verify inverse: " << elapsed.count() << " seconds\n";
    
    hipFree(d_A);

    return 0;
}
