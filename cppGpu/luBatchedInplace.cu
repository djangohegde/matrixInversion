#include "luBatchedInplace.cuh"

int main() {
    int matrixSize, numMatrices, numThreads;

    std::cout << "Enter matrix size: ";
    std::cin >> matrixSize;
    std::cout << "Enter number of matrices: ";
    std::cin >> numMatrices;
    std::cout << "Enter number of threads: ";
    std::cin >> numThreads;
    // numThreads = 30;
    
    int numElements = matrixSize * matrixSize * numMatrices;

    std::vector<FpType> A(numElements);
    std::vector<FpType> A_inv(numElements);

    FpType* d_A;
    hipMalloc(&d_A, numElements * sizeof(FpType));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    FpType inputMatrix[] = {4, 11, 3, 4, 10, 4, 2, 4, 2};

    for (int k = 0; k < numMatrices; ++k) {
        int offset = k * matrixSize * matrixSize;
        std::ifstream file("matrix.txt");
        for (int i = 0; i < matrixSize; ++i) {
            for (int j = 0; j < matrixSize; ++j) {
                // file >> A[(i * matrixSize) + offset + j];
                A[(i * matrixSize) + offset + j] = inputMatrix[(i * matrixSize) + j];
                // A[(i * matrixSize) + offset + j] = rand() % 10;
                A_inv[(i * matrixSize) + offset + j] = 0.0;
            }
        }
        file.close();
    }

    hipMemcpy(d_A, A.data(), numElements * sizeof(FpType), hipMemcpyHostToDevice);
    std::cout << "Data copied to device." << '\n';

    int shMemSize = matrixSize * matrixSize * sizeof(FpType);
    hipFuncSetAttribute(reinterpret_cast<const void*>(batched_lu), hipFuncAttributeMaxDynamicSharedMemorySize, shMemSize);

    hipEventRecord(start, 0);
    batched_lu<<<numMatrices, numThreads, shMemSize>>>(d_A, matrixSize, numMatrices);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " milliseconds\n";

    hipMemcpy(A_inv.data(), d_A, numElements * sizeof(FpType), hipMemcpyDeviceToHost);
    std::cout << "Data copied back to host." << '\n';

    // // print A_inv
    // for (int k = 0; k < numMatrices; ++k) {
    //     int offset = k * matrixSize * matrixSize;
    //     std::cout << "Matrix " << k << ":\n";
    //     for (int i = 0; i < matrixSize; ++i) {
    //         for (int j = 0; j < matrixSize; ++j) {
    //             std::cout << A_inv[(i * matrixSize) + offset + j] << " ";
    //         }
    //         std::cout << '\n';
    //     }
    // }

    auto startT = std::chrono::high_resolution_clock::now();
    verifyInv(A, A_inv, matrixSize, numMatrices);
    auto endT = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = endT - startT;
    std::cout << "Time taken to verify inverse: " << elapsed.count() << " seconds\n";
    
    hipFree(d_A);

    return 0;
}
